#include "hip/hip_runtime.h"
__global__ void vecAdd(float* A, float* B, float* C) {
	int i = threadIdx.x;
	A[i] = 0;
	B[i] = i;
	C[i] = A[i] + B[i];
}

#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 10


void map_on( int x, int y, int* mx, int* my ) {
		*mx = x;
		*my =  *mx + y - 1;
}

void map_off( int mx, int my, int* x, int* y ) {
		*x = mx;
		*y = my - mx + 1;
}





char* strrev(char * string) {
	int length = strlen(string);
	char * result = (char*)malloc(length+1);
	if( result != NULL ) {
		int i,j;                                         
		result[length] = '\0';
		for ( i = length-1, j=0;   i >= 0;   i--, j++ )  
			result[j] = string[i];
	}
	strcpy(string, result);
	free(result);
	
	return string;
}

int read_string(char* filename, char** s, int* len) {
	FILE* fd = fopen( filename, "r" );
	if( fd == NULL ) {
		fprintf( stderr, "Could not open file %s for reading\n", filename );
		return -1;
	}

	fseek(fd, 0, SEEK_END);
	*len = ftell(fd);
	fseek(fd, 0, SEEK_SET);

	*s = (char*)malloc( *len * sizeof(char) );

	fread(*s, *len, 1, fd);

	fclose( fd );
	
	return 0;
}

double get_timeofday() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec+(tv.tv_usec/1000000.0);
}



int parse_args( int argc, char* argv[], char** file1, char** file2, int* maxt, int* maxb ) {
	if( argc < 3 ) {
		return 0;
	}
	
	*file1 = argv[1];
	*file2 = argv[2];
	
	if( maxt != NULL ) {
		if(argc >= 4) {
			*maxt = atoi( argv[3] );
			if( !*maxt ) { 
				return 0; 
			}
		} else {
			*maxt = 32;
		}
	}
	
	if( maxb != NULL ) {
		if(argc >= 5) {
			*maxb = atoi( argv[3] );
			if( !*maxb ) { 
				return 0; 
			}
		} else {
			*maxb = 16;
		}
	}
	
	
	return 1;
}


// grid API

typedef enum { none = 0, up, left, upleft } dir_t;

typedef struct cell_t {
	int len;
	dir_t dir;
} cell_t;

typedef struct grid_t {
	int w;
	int h;
	int d;
	const char* sx;
	const char* sy;
	
	cell_t* cells;
} grid_t;


void grid_init(grid_t* grid, const char* s1, const int m, const char* s2, const int n) {
	grid->sx = s1;
	grid->sy = s2;
	grid->w = m;
	grid->h = n;
	grid->d = grid->w + grid->h - 1;
	
	int mh = grid->d, mw = grid->w;


	size_t size = (mh + 1) * (mw + 1) * sizeof(cell_t);
	grid->cells = (cell_t*)malloc( size );
	
	memset( grid->cells, 0, size );
}


cell_t* grid_cell(const grid_t* grid, const int x, const int y) {

	int mx, my;
	map_on( x, y, &mx, &my );

	return &(grid->cells[ (grid->w + 1)*x + y ]);
}

void lcs_length_cuda( const grid_t* grid, int maxt, int maxb ) {

	int d;
	for(d = 1; d <= grid->d; d++) {
		int x, sx, fx;
		sx = max(d - grid->h + 1, 1);
		fx = min(d, grid->w);
		
		for(x = sx; x <= fx; x++) {
			int i, j;
			map_off( x, d, &i, &j );
		
			//printf( "process [%i, %i] => [%i, %i]\n", d, x, j, i );


			cell_t* cell = grid_cell( grid, i, j );
			//printf("(%i,%i)\n", i, j);

			if(grid->sx[i-1] == grid->sy[j-1]) {
				cell->len = grid_cell( grid, i-1, j-1 )->len + 1;
				cell->dir = upleft;
				
				//printf("ul: [%i, %i] = %c\n", j, i, grid->sx[i-1]);
			} else {
				cell_t* cell_up = grid_cell( grid, i-1, j );
				cell_t* cell_left = grid_cell( grid, i, j-1 );

				if(cell_up->len >= cell_left->len) {
					cell->len = cell_up->len;
					cell->dir = up;
				} else {
					cell->len = cell_left->len;
					cell->dir = left;
				}
			}	
		}

	}

}



void lcs_backtrack_cuda(const grid_t* grid, int i, int j, const int maxlen, char* res, int maxt, int maxb) {

	int pos = 0;
	while(i > 0 && j > 0) {
		cell_t* cell = grid_cell(grid, i, j);
		
		if( cell->dir == upleft ) {
			//printf(":%c @ %i/%i\n", grid->s1[i-1], i, j);
			res[pos++] = grid->sx[i-1];
			i--; j--;
		} else if( cell->dir == up ) {
			i--;
		} else if( cell->dir == left) {
			j--;
		} else {
			//printf("\n---WTF BAD CELL!--\n");
		}
	}
	res[pos] = 0;
	
	strrev(res);
}



void lcs_cuda(const char* s1, const int m, const char* s2, const int n, const int maxlen, char* res, int maxt, int maxb) {
	grid_t grid;
	grid_init( &grid, s1, m, s2, n );

	lcs_length_cuda( &grid, maxt, maxb );
	lcs_backtrack_cuda(&grid, grid.w, grid.h, maxlen, res, maxt, maxb);
}

int main(int argc, char* argv[]) {
	char* s1;
	char* s2;
	int m, n;
	
	
	//s1="MZJAWXU";
	//s2="XMJYAUZ";
	//m = strlen(s1);
	//n = strlen(s2);
	
	char* file1;
	char* file2;
	int maxt, maxb;
	if( !parse_args( argc, argv, &file1, &file2, &maxt, &maxb ) ) {
		printf("Usage: %s file1.txt file2.txt [maxt] [maxb]\n", argv[0]);
		exit(-1);
	}
	
	read_string(file1, &s1, &m);
	read_string(file2, &s2, &n);
	
	
	

	
	// allocate a result string on the device.
	int reslen;
	char* res;
	reslen = max(m, n) + 1;
	res = (char*)malloc( reslen*sizeof(char) );

	double start = get_timeofday();

	lcs_cuda( s1, m, s2, n, reslen, res, maxt, maxb );

	double end = get_timeofday();

	printf("Time Taken: %f sec %i %s\n", end-start, strlen(res), res);
}
