#include "hip/hip_runtime.h"
__global__ void vecAdd(float* A, float* B, float* C) {
	int i = threadIdx.x;
	A[i] = 0;
	B[i] = i;
	C[i] = A[i] + B[i];
}

#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 10

__device__ __host__
void map_on( int x, int y, int* mx, int* my ) {
		*mx = x;
		*my =  x + y - 1;
}

__device__ __host__
void map_off( int mx, int my, int* x, int* y ) {
		*x = mx;
		*y = my - mx + 1;
}






/*
void transform(int m, int n) {
	// alloc d x n grid
	// copy each (x,y) => (d-y+1, d-x+1)

	int num_diag = m + n - 1;
	int nm = num_diag, nn = n;
	
	printf("tranforming %i x %i to %i x %i\n", m, n, nm, nn);

	int x, y;
	for(y = 1; y <= m; y++) {
		for(x = 1; x <= n; x++) {
			int mx, my;
			map_on( x, y, &mx, &my );
			
			//int ox = 0;
			//int oy = 0;
			//map_off( nx, ny, &ox, &oy );
			
			printf("(%i, %i) => (%i, %i) => (%i, %i)\n", x, y, mx, my);
		}
	}
}
*/


char* strrev(char * string) {
	int length = strlen(string);
	char * result = (char*)malloc(length+1);
	if( result != NULL ) {
		int i,j;                                         
		result[length] = '\0';
		for ( i = length-1, j=0;   i >= 0;   i--, j++ )  
			result[j] = string[i];
	}
	strcpy(string, result);
	free(result);
	
	return string;
}

int read_string(char* filename, char** s, int* len) {
	FILE* fd = fopen( filename, "r" );
	if( fd == NULL ) {
		fprintf( stderr, "Could not open file %s for reading\n", filename );
		return -1;
	}

	fseek(fd, 0, SEEK_END);
	*len = ftell(fd);
	fseek(fd, 0, SEEK_SET);

	*s = (char*)malloc( *len * sizeof(char) );

	fread(*s, *len, 1, fd);

	fclose( fd );
	
	return 0;
}

double get_timeofday() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec+(tv.tv_usec/1000000.0);
}


// grid API

typedef enum { none = 0, up, left, upleft } dir_t;

typedef struct cell_t {
	int len;
	dir_t dir;
} cell_t;

typedef struct grid_t {
	int w;
	int h;
	int d;
	char* sx;
	char* sy;
	
	cell_t* cells;
} grid_t;


void cudaMallocOrDie( void** pptr, const size_t size, const char* code ) {
	if( hipMalloc(pptr, size) != hipSuccess) {
		printf("hipMalloc Failed: %s\n", code);
		exit(-1);
	}
}

void cudaMemcpyOrDie( void* dst, const void* src, const size_t size, const enum hipMemcpyKind kind, const char* code ) {
	if( hipMemcpy( dst, src, size, kind ) != hipSuccess) {
		printf("hipMalloc Failed: %s\n", code);
		exit(-1);
	}
}

void cudaMemsetOrDie( void* dst, char value, size_t size, const char* code ) {
	if( hipMemset( dst, value, size ) != hipSuccess) {
		printf("hipMemset Failed: %s\n", code);
		exit(-1);
	}
}

void grid_init(grid_t* dgrid, const char* s1, const int m, const char* s2, const int n) {
	grid_t grid;

	// set sizes
	grid.w = m;
	grid.h = n;
	grid.d = grid.w + grid.h - 1;
	
	// copy strings to device
	cudaMallocOrDie( (void**)&(grid.sx), m*sizeof(char), "grid_init 1" );
	cudaMemcpyOrDie( grid.sx, s1, m*sizeof(char), hipMemcpyHostToDevice, "grid_init 2" );
	
	cudaMallocOrDie( (void**)&(grid.sy), n*sizeof(char), "grid_init 3" );
	cudaMemcpyOrDie( grid.sy, s2, n*sizeof(char), hipMemcpyHostToDevice, "grid_init 4" );

	// create cell grid on device
	size_t size = (grid.d + 1) * (grid.w + 1) * sizeof(cell_t);
	printf("size=%i\n", size);
	cudaMallocOrDie( (void**)&(grid.cells), size, "grid_init 5" );
	cudaMemsetOrDie( grid.cells, 0, size, "grid_init 6" );
	
	// copy the stuff we just created to the device. NOTE all pointers
	// are relative to device.
	cudaMemcpyOrDie( dgrid, &grid, sizeof(grid_t), hipMemcpyHostToDevice, "grid_init 7" );
}

__device__ 
cell_t* grid_cell(const grid_t* grid, const int x, const int y) {

	int mx, my;
	map_on( x, y, &mx, &my );

	return &(grid->cells[ (grid->w + 1)*x + y ]);
}

/*
void build_table(int m, int n) {
	int num_diag = m + n - 1;
	
	int x, y;
	for(y = 1; y <= m; y++) {
		for(x = 1; x <= n; x++) {
			int mx, my;
			
			// get neighboring values
			map_on( x, y - 1, &mx, &my );
			int val_up = grid( mx, my );
			map_on( x - 1, y, &mx, &my );
			int val_left = grid( mx, my );
			map_on( x - 1, y - 1, &mx, &my );
			int val_upleft = grid( mx, my );
			
			// set value of cell
			int val;
			if( s2[x-1] == s1[y-1] ) {
				val = val_upleft + 1;
			} else {
				val = max( val_up, val_left );
			}

			map_on( x, y, &mx, &my );
			grid( mx, my ) = val;
			
		
			// same LCS build table code, but with nx and ny mapping.
			// x = thread. maybe x/p loop.
		}
	}
}

*/



/*

Computes a block of the LCS length matrix. Before the call, 
all cells immediately above and to the left of the block of interest 
must be calculated. that is, the cells above as ([x, x+w], y-1) and
to the left as ([y, y+h], x-1). These are the cells represented by #.

                  x       x+w
  +-------+-------+-------+-------+
  | * * * | * * * | * * * |       |
  | * * * | * * # | # # # |       |
  +-------+-------+-------+-------+ y
  | * * * | * * # | 0 0 0 |       |
  | * * * | * * # | 0 0 0 |       |
  +-------+-------+-------+-------+ y+h
  |       |       |       |       |
  |       |       |       |       |
  +-------+-------+-------+-------+
  
This structure allows for usage by either message passing or shared memory based
formulations.
  
*/


__global__
void lcs_length_row_cuda( const grid_t* grid, int d, int bw, int bh ) {

	int x = blockDim.x * blockIdx.x + threadIdx.x;
	
	int y;
	for(y = d; y < d + bh; y++) {
		int i, j;
		map_off( x, y, &i, &j );
	
		//printf( "process [%i, %i] => [%i, %i]\n", d, x, j, i );


		cell_t* cell = grid_cell( grid, i, j );
		//printf("(%i,%i)\n", i, j);

		if(grid->sx[i-1] == grid->sy[j-1]) {
			cell->len = grid_cell( grid, i-1, j-1 )->len + 1;
			cell->dir = upleft;
			
			//printf("ul: [%i, %i] = %c\n", j, i, grid->sx[i-1]);
		} else {
			cell_t* cell_up = grid_cell( grid, i-1, j );
			cell_t* cell_left = grid_cell( grid, i, j-1 );

			if(cell_up->len >= cell_left->len) {
				cell->len = cell_up->len;
				cell->dir = up;
			} else {
				cell->len = cell_left->len;
				cell->dir = left;
			}
		}	
	}
}


void lcs_length_cuda( const grid_t* dgrid, int maxb, int maxt ) {

	// FIXME
	grid_t lgrid;
	grid_t* grid = &lgrid;
	cudaMemcpyOrDie( grid, dgrid, sizeof(grid_t), hipMemcpyDeviceToHost, "lcs_length_cuda 1" );


	int bw = ceil(grid->w / (double)maxb);
	int bh = 10;

	int nd = grid->d/bh;

	int d;
	for(d = 0; d < nd; d ++) {
	
		int sx = max(d - grid->h, 0);
		int fx = min(d, grid->w - 1);
		
		int nt = min(bw, maxt);
		int nb = min(ceil((fx - sx)/ (double)bw), (double)maxb);
		
		
		printf("sx=%i, fx=%i, nb=%i, nt=%i, bw=%i, bh=%i\n", sx, fx, nb, nt, bw, bh);

		
		//lcs_length_row_cuda<<<nb, nt>>>( dgrid, d, bw, bh );
	}

}



__global__
void lcs_backtrack_cuda(const grid_t* grid, int i, int j, const int maxlen, char* res) {

	int pos = 0;
	while(i > 0 && j > 0) {
		cell_t* cell = grid_cell(grid, i, j);
		
		if( cell->dir == upleft ) {
			//printf(":%c @ %i/%i\n", grid->s1[i-1], i, j);
			res[pos++] = grid->sx[i-1];
			i--; j--;
		} else if( cell->dir == up ) {
			i--;
		} else if( cell->dir == left) {
			j--;
		} else {
			//printf("\n---WTF BAD CELL!--\n");
		}
	}
	res[pos] = 0;
}



void lcs_cuda(const char* s1, const int m, const char* s2, const int n, const int maxlen, char* res) {
	
	// create grid on device
	grid_t* grid;
	cudaMallocOrDie( (void**)&grid, sizeof(grid_t), "lcs_cuda 1");
	grid_init( grid, s1, m, s2, n );


	// run length algorithm (makes kernel calls)
	lcs_length_cuda( grid, 20, 32 );
	
	// create a string on the device for result, and calculate result
	char* dres;
	cudaMallocOrDie( (void**)&dres, maxlen*sizeof(char), "lcs_cuda 2");

	// FIXME use grid h/w 
	//lcs_backtrack_cuda<<<1, 1>>>(grid, m, n, maxlen, dres);
	
	// copy result back, and reverse it (calculated backwards).
	cudaMemcpyOrDie( res, dres, maxlen, hipMemcpyDeviceToHost, "lcs_cuda 3" );

	strrev(res);
}

int main() {
	char* s1;
	char* s2;
	int m, n;
	
	
	s1="MZJAWXUMZJAWXUMZJAWXUMZJAWXU";
	s2="XMJYAUZXMJYAUZXMJYAUZXMJYAUZ";
	m = strlen(s1);
	n = strlen(s2);
	
	read_string("small_test1.txt", &s1, &m);
	read_string("small_test2.txt", &s2, &n);
	

	
	// allocate a result string on the device.
	int reslen;
	char* res;
	reslen = max(m, n) + 1;
	res = (char*)malloc( reslen*sizeof(char) );

	double start = get_timeofday();

	lcs_cuda( s1, m, s2, n, reslen, res );

	double end = get_timeofday();

	printf("Time Taken: %f sec %i %s\n", end-start, strlen(res), res);
}

/*


int main() {
	char* s1;
	char* s2;
	int m, n;
	
	
	//s1="MZJAWXU";
	//s2="XMJYAUZ";
	//m = strlen(s1);
	//n = strlen(s2);
	
	read_string("large_test1.txt", &s1, &m);
	read_string("large_test2.txt", &s2, &n);
	
	// copy strings to device
	char* ds1;
	char* ds2;
	hipMalloc( &ds1, m );
	hipMalloc( &ds2, n );
	hipMemcpy( ds1, s1, m, hipMemcpyHostToDevice );
	hipMemcpy( ds2, s2, n, hipMemcpyHostToDevice );
	
	// allocate space for the grid on the device
	int d = m + n - 1;
	int mh = d, mw = m;
	size_t size = (mh + 1) * (mw + 1) * sizeof(cell_t);
	cell_t* cells;
	hipMalloc( &cells, size );
	hipMemset( cells, 0, size );
	
	// allocate a result string on the device.
	int reslen;
	char* res;
	reslen = max(m, n) + 1;
	hipMalloc(&res, reslen*sizeof(char));

	double start = get_timeofday();

	lcs_cuda( ds1, m, ds2, n, reslen, res );

	double end = get_timeofday();
	
	char* rres = (char*)malloc(reslen);
	
	hipMemcpy( rres, res, reslen, hipMemcpyDeviceToHost );
	
	printf("Time Taken: %f sec %i %s\n", end-start, strlen(rres), rres);
	

}

*/
